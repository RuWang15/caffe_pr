#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <>
void caffe_gpu_gemm_ipdf<float>(const int N, const int M, const int K, 
      const float* A, const float* B, float* C, const bool trans, 
      const int T, const int start) {
  // Note that cublas follows fortran order.
  float *Bt;
  float alpha = 1.0, beta = 0.0;
  hipError_t memStat = hipMalloc((void**)&Bt, M*K*sizeof(float));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Bt";
  // calculate B^T
  hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, &alpha, B, K, &beta, B, K, Bt, M);
  
  if (trans) {
    // A: weight(NxK) * B: bottom(KxM) = C: top(NxM)
    // calculate A * B
    hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, 
      N, M, 1+(K-start-1)/T, &alpha, A+start*N, N*T, Bt+start*M, M*T, &beta, C, N);
  } else { 
    // (A: weight(KxN))^T * B: bottom(KxM) = C: top(NxM)
    // calculate A^T
    float *At;
    memStat = hipMalloc((void**)&At, N*K*sizeof(float));
    if (memStat != hipSuccess) 
      LOG(ERROR) << "hipMalloc Failed! At";
    hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, &alpha, A, K, &beta, A, K, At, N);
    // calculate A^T * B
    hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, 
      N, M, 1+(K-start-1)/T, &alpha, At+start*N, N*T, Bt+start*M, M*T, &beta, C, N);
    hipFree(At);
  }
  hipFree(Bt);
}

template <>
void caffe_gpu_gemm_ipdf<double>(const int N, const int M, const int K, 
      const double* A, const double* B, double* C, const bool trans, 
      const int T, const int start) {
  // Note that cublas follows fortran order.
  double *Bt;
  double alpha = 1.0, beta = 0.0;
  hipError_t memStat = hipMalloc((void**)&Bt, M*K*sizeof(double));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Bt";
  // calculate B^T
  hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, &alpha, B, K, &beta, B, K, Bt, M);
  
  if (trans) {
    // A: weight(NxK) * B: bottom(KxM) = C: top(NxM)
    // calculate A * B
    hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, 
      N, M, 1+(K-start-1)/T, &alpha, A+start*N, N*T, Bt+start*M, M*T, &beta, C, N);
  } else { 
    // (A: weight(KxN))^T * B: bottom(KxM) = C: top(NxM)
    // calculate A^T
    double *At;
    memStat = hipMalloc((void**)&At, N*K*sizeof(double));
    if (memStat != hipSuccess) 
      LOG(ERROR) << "hipMalloc Failed! At";
    hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, &alpha, A, K, &beta, A, K, At, N);
    // calculate A^T * B
    hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_T, 
      N, M, 1+(K-start-1)/T, &alpha, At+start*N, N*T, Bt+start*M, M*T, &beta, C, N);
    hipFree(At);
  }
  hipFree(Bt);
}

template <>
void caffe_gpu_gemm_ipdb_w<float>(const int N, const int M, const int K, 
      const float* A, const float* B, float* C, const bool trans, 
      const int T, const int start) {
  // A: top_diff(NxM); B: bottom_data(KxM)
  float* Bt;
  hipError_t memStat = hipMalloc((void**)&Bt, M*K*sizeof(float));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Bt";
  float alpha = 1.0, beta1 = 0.0, beta2 = 1.0;;
  // Bt = (B: bottom_data(KxM))^T
  hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, 
    &alpha, B, K, &beta1, B, K, Bt, M);
  if (trans) {
    // C(NxK) = A: top_diff(NxM) * Bt: (bottom_data(KxM))^T + C(NxK)
    hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1+(K-start-1)/T, M,
      &alpha, A, N, Bt+start*M, M*T, &beta2, C+start*N, N*T);
  } else {
    float* Ct;
    memStat = hipMalloc((void**)&Ct, N*K*sizeof(float));
    if (memStat != hipSuccess) 
      LOG(ERROR) << "hipMalloc Failed! Ct";
    // Ct(NxK) = C^T
    hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, 
      &alpha, C, K, &beta1, C, K, Ct, N);
    // Ct(NxK) = A: top_diff(NxM) * Bt: (bottom_data(KxM))^T + Ct(NxK)
    hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1+(K-start-1)/T, M,
      &alpha, A, N, Bt+start*M, M*T, &beta2, Ct+start*N, N*T);
    // C(KxN) = Ct^T
    hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, N, 
      &alpha, Ct, N, &beta1, Ct, N, C, K);
    hipFree(Ct);
  }
  hipFree(Bt);
}

template <>
void caffe_gpu_gemm_ipdb_w<double>(const int N, const int M, const int K, 
      const double* A, const double* B, double* C, const bool trans, 
      const int T, const int start) {
  // A: top_diff(NxM); B: bottom_data(KxM)
  double* Bt;
  hipError_t memStat = hipMalloc((void**)&Bt, M*K*sizeof(double));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Bt";
  double alpha = 1.0, beta1 = 0.0, beta2 = 1.0;;
  // Bt = (B: bottom_data(KxM))^T
  hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, 
    &alpha, B, K, &beta1, B, K, Bt, M);
  if (trans) {
    // C(NxK) = A: top_diff(NxM) * Bt: (bottom_data(KxM))^T + C(NxK)
    hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1+(K-start-1)/T, M,
      &alpha, A, N, Bt+start*M, M*T, &beta2, C+start*N, N*T);
  } else {
    double* Ct;
    memStat = hipMalloc((void**)&Ct, N*K*sizeof(double));
    if (memStat != hipSuccess) 
      LOG(ERROR) << "hipMalloc Failed! Ct";
    // Ct(NxK) = C^T
    hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, 
      &alpha, C, K, &beta1, C, K, Ct, N);
    // Ct(NxK) = A: top_diff(NxM) * Bt: (bottom_data(KxM))^T + Ct(NxK)
    hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_N, HIPBLAS_OP_N, N, 1+(K-start-1)/T, M,
      &alpha, A, N, Bt+start*M, M*T, &beta2, Ct+start*N, N*T);
    // C(KxN) = Ct^T
    hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, N, 
      &alpha, Ct, N, &beta1, Ct, N, C, K);
    hipFree(Ct);
  }
  hipFree(Bt);
}

template <>
void caffe_gpu_gemm_ipdb_bd<float>(const int N, const int M, const int K, 
      const float* A, const float* B, float* C, const bool trans, 
      const int T, const int start) {
	// A: top_diff(NxM); B: weight
	float alpha = 1.0, beta = 0.0;
	float* Ct;
	hipError_t memStat = hipMalloc((void**)&Ct, M*K*sizeof(float));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Ct";
  memStat = hipMemset(Ct, 0, M*K*sizeof(float));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMemset Failed! Ct";
	if (trans) { // B: weight(NxK)
		// Ct(MxK) = (A: top_diff(NxM))^T * B: weight(NxK)
		hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, M, 1+(K-start-1)/T, N,
      &alpha, A, N, B+start*N, N*T, &beta, Ct+start*M, M*T);
		// C(KxM) = Ct^T
		hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, M, 
      &alpha, Ct, M, &beta, Ct, M, C, K);
	} else { // B: weight(KxN)
		float* Bt;
		memStat = hipMalloc((void**)&Bt, N*K*sizeof(float));
	  if (memStat != hipSuccess) 
	    LOG(ERROR) << "hipMalloc Failed! Bt";
	  // Bt(NxK) = B^T
	  hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, 
      &alpha, B, K, &beta, B, K, Bt, N);
	  // Ct(MxK) = (A: top_diff(NxM))^T * Bt(NxK)
	  hipblasSgemm(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, M, 1+(K-start-1)/T, N,
      &alpha, A, N, Bt+start*N, N*T, &beta, Ct+start*M, M*T);
		// C(KxM) = Ct^T
		hipblasSgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, M, 
      &alpha, Ct, M, &beta, Ct, M, C, K);
		hipFree(Bt);
	}
	hipFree(Ct);
}

template <>
void caffe_gpu_gemm_ipdb_bd<double>(const int N, const int M, const int K, 
      const double* A, const double* B, double* C, const bool trans, 
      const int T, const int start) {
	// A: top_diff(NxM); B: weight
	double alpha = 1.0, beta = 0.0;
	double* Ct;
	hipError_t memStat = hipMalloc((void**)&Ct, M*K*sizeof(double));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMalloc Failed! Ct";
  memStat = hipMemset(Ct, 0, M*K*sizeof(double));
  if (memStat != hipSuccess) 
    LOG(ERROR) << "hipMemset Failed! Ct";
	if (trans) { // B: weight(NxK)
		// Ct(MxK) = (A: top_diff(NxM))^T * B: weight(NxK)
		hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, M, 1+(K-start-1)/T, N,
      &alpha, A, N, B+start*N, N*T, &beta, Ct+start*M, M*T);
		// C(KxM) = Ct^T
		hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, M, 
      &alpha, Ct, M, &beta, Ct, M, C, K);
	} else { // B: weight(KxN)
		double* Bt;
		memStat = hipMalloc((void**)&Bt, N*K*sizeof(double));
	  if (memStat != hipSuccess) 
	    LOG(ERROR) << "hipMalloc Failed! Bt";
	  // Bt(NxK) = B^T
	  hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, N, K, 
      &alpha, B, K, &beta, B, K, Bt, N);
	  // Ct(MxK) = (A: top_diff(NxM))^T * Bt(NxK)
	  hipblasDgemm(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_N, M, 1+(K-start-1)/T, N,
      &alpha, A, N, Bt+start*N, N*T, &beta, Ct+start*M, M*T);
		// C(KxM) = Ct^T
		hipblasDgeam(Caffe::cublas_handle(), HIPBLAS_OP_T, HIPBLAS_OP_T, K, M, 
      &alpha, Ct, M, &beta, Ct, M, C, K);
		hipFree(Bt);
	}
	hipFree(Ct);
}

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe

